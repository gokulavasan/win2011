#include "hip/hip_runtime.h"

#include "ImageCleaner.h"

//YOU CAN CHANGE THESE TO MATCH YOUR IMAGE SIZE
#define SIZEX    512
#define SIZEY    512

#define PI 3.1415926536
#define BLOCK_SIZE 512
//----------------------------------------------------------------
// TODO:  CREATE NEW KERNELS HERE.  YOU CAN PLACE YOUR CALLS TO
//        THEM IN THE INDICATED SECTION INSIDE THE 'filterImage'
//        FUNCTION.
//
// BEGIN ADD KERNEL DEFINITIONS
//----------------------------------------------------------------

__device__ float GetElement(const float *image, const int row, int col)
{
    return image[row * SIZEX + col];
}

__device__ void SetElement(float *image, const int row, int col, float val)
{
    image[row * SIZEX + col] = val;
}



// This is an example kernel defintion that you should consider using
__global__ void DFTKernel (float *real_image, float *imag_image,
                            int size_x, int size_y, int direction, int forward)
{
  
  int blockId = blockIdx.x;
  int threadId = threadIdx.x;


  float real_Xvalue[SIZEX/BLOCK_SIZE];
  float imag_Xvalue[SIZEX/BLOCK_SIZE];
  __shared__ float real_vect[BLOCK_SIZE];
  __shared__ float imag_vect[BLOCK_SIZE];

  for (int z = 0; z < SIZEX/BLOCK_SIZE ; ++z){
    real_Xvalue[z] = 0;
    imag_Xvalue[z] = 0;
  }

  for (int i = 0; i < SIZEX/BLOCK_SIZE; ++i){
    
    if (direction == 0) {
      // Row direction 
      real_vect[threadId] = GetElement(real_image, blockId, threadId + i*BLOCK_SIZE);
      imag_vect[threadId] = GetElement(imag_image, blockId, threadId + i*BLOCK_SIZE);
    }  else {
      // Col direction
      real_vect[threadId] = GetElement(real_image, threadId + i*BLOCK_SIZE, blockId);
      imag_vect[threadId] = GetElement(imag_image, threadId + i*BLOCK_SIZE, blockId);
    }
  
    __syncthreads();
    
    // need to calculate the 'theta' value, based on thread id, 
    //   and also different for forward and reverse
    float theta = 0;
    for (int z = 0; z < SIZEX/BLOCK_SIZE; ++z){
      for (int e = 0; e < BLOCK_SIZE; ++e){
        if (forward == 1) {
          // Forward DFT
          theta = -2*PI*threadId*(e+z*BLOCK_SIZE)/SIZEX;
        } else {
          // Inverse DFT
          theta = 2*PI*threadId*(e+z*BLOCK_SIZE)/SIZEX;
        }
        real_Xvalue[z] += real_vect[e]*cosf(theta)-imag_vect[e]*sinf(theta);
        imag_Xvalue[z] += imag_vect[e]*cosf(theta)+real_vect[e]*sinf(theta);
      }
    }
    __syncthreads();
  }

  for (int z = 0; z < SIZEX/BLOCK_SIZE ;++z){
    if (forward != 1){
      real_Xvalue[z] /= SIZEX;
      imag_Xvalue[z] /= SIZEX;
    }
    if (forward == 1 && direction ==1) {
      const int eightX = SIZEX/8;
      const int eight7X = SIZEX - eightX;
      const int eightY = SIZEY/8;
      const int eight7Y = SIZEY - eightY;
  
      int x = threadId+z*BLOCK_SIZE;
      int y = blockId;
  
      if (!(x < eightX && y < eightY) &&
          !(x < eightX && y >= eight7Y) &&
          !(x >= eight7X && y < eightY) &&
          !(x >= eight7X && y >= eight7Y))
         {
         real_Xvalue[z]=0;
         imag_Xvalue[z]=0;
      }
    }

    if (direction ==0){
      SetElement(real_image, blockId, threadId+z*BLOCK_SIZE, real_Xvalue[z]);
      SetElement(imag_image, blockId, threadId+z*BLOCK_SIZE, imag_Xvalue[z]);
    } else {
      SetElement(real_image, threadId+z*BLOCK_SIZE, blockId, real_Xvalue[z]);
      SetElement(imag_image, threadId+z*BLOCK_SIZE, blockId, imag_Xvalue[z]);
    }
  }

}

//----------------------------------------------------------------
// END ADD KERNEL DEFINTIONS
//----------------------------------------------------------------

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // check that the sizes match up
  assert(size_x == SIZEX);
  assert(size_y == SIZEY);

  int matSize = size_x * size_y * sizeof(float);

  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;
  CUDA_ERROR_CHECK(hipEventCreate(&start));
  CUDA_ERROR_CHECK(hipEventCreate(&stop));

  // Create a stream and initialize it
  hipStream_t filterStream;
  CUDA_ERROR_CHECK(hipStreamCreate(&filterStream));

  // Alloc space on the device
  float *device_real, *device_imag;
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_real, matSize));
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_imag, matSize));

  // Start timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));
  
  // Here is where we copy matrices down to the device 
  CUDA_ERROR_CHECK(hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice));
  CUDA_ERROR_CHECK(hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice));
  
  // Stop timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferDown,start,stop));

  // Start timing for the execution
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));
 
  //----------------------------------------------------------------
  // TODO:  YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  //
  // BEGIN ADD KERNEL CALLS
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual and learn about dynamically allocating shared memory
  //    4. Stream to execute kernel on, should always be 'filterStream'
  //
  // Also note that you pass the pointers to the device memory to the kernel call
  DFTKernel<<<size_y,BLOCK_SIZE,0,filterStream>>>(device_real,device_imag,size_x,size_y,0,1);
  DFTKernel<<<size_x,BLOCK_SIZE,0,filterStream>>>(device_real,device_imag,size_x,size_y,1,1);


 
  DFTKernel<<<size_y,BLOCK_SIZE,0,filterStream>>>(device_real,device_imag,size_x,size_y,0,0);
  DFTKernel<<<size_x,BLOCK_SIZE,0,filterStream>>>(device_real,device_imag,size_x,size_y,1,0);



  //---------------------------------------------------------------- 
  // END ADD KERNEL CALLS
  //----------------------------------------------------------------
  // Finish timimg for the execution 
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&execution,start,stop));

  // Start timing for the transfer up
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  // Here is where we copy matrices back from the device 
  CUDA_ERROR_CHECK(hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost));

  // Finish timing for transfer up
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferUp,start,stop));

  // Synchronize the stream
  CUDA_ERROR_CHECK(hipStreamSynchronize(filterStream));
  // Destroy the stream
  CUDA_ERROR_CHECK(hipStreamDestroy(filterStream));
  // Destroy the events
  CUDA_ERROR_CHECK(hipEventDestroy(start));
  CUDA_ERROR_CHECK(hipEventDestroy(stop));

  // Free the memory
  CUDA_ERROR_CHECK(hipFree(device_real));
  CUDA_ERROR_CHECK(hipFree(device_imag));

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
  // Return the total time to transfer and execute
  return totalTime;
}

